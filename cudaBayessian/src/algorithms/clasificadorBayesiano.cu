#include "hip/hip_runtime.h"
#include <iostream>
#include <core.cuh>
#include <kernels.cuh>
void clasificarBayesiano(const float *h_test_data,
                         const ColumnStats *h_stats_pos,
                         const ColumnStats *h_stats_neg,
                         float prior_pos,
                         float prior_neg,
                         int *h_predictions,
                         float *h_log_likelihood_pos,
                         float *h_log_likelihood_neg,
                         int n_test,
                         int cols,
                         int target_col_index,
                         float epsilon)
{

    // Calcular log priors
    float log_prior_pos = logf(prior_pos);
    float log_prior_neg = logf(prior_neg);

    // Asignar memoria en GPU
    float *d_test_data;
    ColumnStats *d_stats_pos, *d_stats_neg;
    int *d_predictions;
    float *d_log_likelihood_pos, *d_log_likelihood_neg;

    hipMalloc(&d_test_data, n_test * cols * sizeof(float));
    hipMalloc(&d_stats_pos, cols * sizeof(ColumnStats));
    hipMalloc(&d_stats_neg, cols * sizeof(ColumnStats));
    hipMalloc(&d_predictions, n_test * sizeof(int));
    hipMalloc(&d_log_likelihood_pos, n_test * sizeof(float));
    hipMalloc(&d_log_likelihood_neg, n_test * sizeof(float));

    // Copiar datos a GPU
    hipMemcpy(d_test_data, h_test_data, n_test * cols * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(d_stats_pos, h_stats_pos, cols * sizeof(ColumnStats),
               hipMemcpyHostToDevice);
    hipMemcpy(d_stats_neg, h_stats_neg, cols * sizeof(ColumnStats),
               hipMemcpyHostToDevice);

    // Configurar kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n_test + threadsPerBlock - 1) / threadsPerBlock;

    // Ejecutar kernel
    naiveBayesKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_test_data,
        d_stats_pos,
        d_stats_neg,
        log_prior_pos,
        log_prior_neg,
        d_predictions,
        d_log_likelihood_pos,
        d_log_likelihood_neg,
        n_test,
        cols,
        target_col_index,
        epsilon);

    // Verificar errores
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "Error en kernel: " << hipGetErrorString(err) << std::endl;
    }

    hipDeviceSynchronize();

    // Copiar resultados de vuelta
    hipMemcpy(h_predictions, d_predictions, n_test * sizeof(int),
               hipMemcpyDeviceToHost);
    hipMemcpy(h_log_likelihood_pos, d_log_likelihood_pos, n_test * sizeof(float),
               hipMemcpyDeviceToHost);
    hipMemcpy(h_log_likelihood_neg, d_log_likelihood_neg, n_test * sizeof(float),
               hipMemcpyDeviceToHost);

    // Liberar memoria GPU
    hipFree(d_test_data);
    hipFree(d_stats_pos);
    hipFree(d_stats_neg);
    hipFree(d_predictions);
    hipFree(d_log_likelihood_pos);
    hipFree(d_log_likelihood_neg);
}